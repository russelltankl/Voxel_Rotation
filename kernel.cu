#include "hip/hip_runtime.h"
// TestMatrix.cpp : Defines the entry point for the console application.
//

#include "stdafx.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include "FloatVector.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define PI 3.14159265
using namespace std;

// Multiply a 3 x 3 Matrix size
void multiply3x3(Float3 *firstMat, Float3 *secondMat, Float3 *outputMat)
{
	int i, j, k;
	for (i = 0; i < 3; i++)
	{
		for (j = 0; j < 3; j++)
		{
			for (k = 0; k < 3; k++)
				outputMat[j].value[i] += firstMat[k].value[i] * secondMat[j].value[k];
		}
	}
	cout << "Printing a 3 x 3 Matrix: " << endl;
	for (i = 0; i < 3; i++) {
		for (j = 0; j < 3; j++) {
			cout << outputMat[j].value[i] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

// Multiply a 4 x 4 Matrix size
void multiply4x4(Float4 *firstMat, Float4 *secondMat, Float4 *outputMat)
{
	int i, j, k;
	for (i = 0; i < 4; i++)
	{
		for (j = 0; j < 4; j++)
		{
			for (k = 0; k < 4; k++)
				outputMat[j].value[i] += firstMat[k].value[i] * secondMat[j].value[k];
		}
	}
	//cout << "Printing a 4 x 4 Matrix: " << endl;
	//for (i = 0; i < 4; i++) {
	//	for (j = 0; j < 4; j++) {
	//		cout << outputMat[j].value[i] << " ";
	//	}
	//	cout << endl;
	//}
	//cout << endl;
}

// Multiply a n x m Matrix size, where you can define N and M
void multiplynxm(Float4 *firstMat, Float4 *secondMat, Float4 *outputMat, int N, int M)
{
	int i, j, k;
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < M; j++)
		{
			for (k = 0; k < N; k++)
				outputMat[j].value[i] += firstMat[k].value[i] * secondMat[j].value[k];
		}
	}
	//cout << "Transformation Matrix X Location Matrix: " << endl;
	//for (i = 0; i < N; i++) {
	//	for (j = 0; j < M; j++) {
	//		cout << outputMat[j].value[i] << " ";
	//	}
	//	cout << endl;
	//}
	//cout << endl;
}

// Generate a 3 x 3 Transformation Matrix, where you input the rotation axis, coordinate and angle of rotation
Float3 *gen3x3tm(float x, float y, float theta)
{
	Float3 *a = new Float3[3]();
	Float3 *b = new Float3[3]();
	Float3 *c = new Float3[3]();
	Float3 *result1 = new Float3[3];
	Float3 *result2 = new Float3[3];


	a[0].value[0] = 1;
	a[1].value[1] = 1;
	a[2].value[0] = x;
	a[2].value[1] = y;
	a[2].value[2] = 1;

	b[0].value[0] = 1;
	b[1].value[1] = 1;
	b[2].value[0] = -x;
	b[2].value[1] = -y;
	b[2].value[2] = 1;

	c[0].value[0] = floor(cos(theta*PI / 180));
	c[0].value[1] = floor(sin(theta*PI / 180));
	c[1].value[0] = floor(-sin(theta*PI / 180));
	c[1].value[1] = floor(cos(theta*PI / 180));
	c[2].value[2] = 1;

	multiply3x3(a, c, result1);
	multiply3x3(result1, b, result2);
	delete[](a);
	delete[](b);
	delete[](c);
	delete[](result1);

	return result2;
}

// Generate a 4 x 4 Transformation Matrix, where you input the rotation axis, coordinate and angle of rotation
Float4 *gen4x4tm(float xCoor, float yCoor, float zCoor, char axis, float theta)
{
	Float4 *a = new Float4[4]();
	Float4 *b = new Float4[4]();
	Float4 *c = new Float4[4]();
	Float4 *result1 = new Float4[4];
	Float4 *result2 = new Float4[4];

	a[0].value[0] = 1;
	a[1].value[1] = 1;
	a[2].value[2] = 1;
	a[3].value[0] = xCoor;
	a[3].value[1] = yCoor;
	a[3].value[2] = zCoor;
	a[3].value[3] = 1;

	b[0].value[0] = 1;
	b[1].value[1] = 1;
	b[2].value[2] = 1;
	b[3].value[3] = 1;

	c[3].value[3] = 1;

	// TEMPORARY FIX
	int OddEven = 1;
	if (OddEven == 1)
	{
		float Divisable = 32.0 / 2.0; //16
		float Divisable1 = Divisable - 1.0; //15

		if (axis == 'x')
		{
			c[0].value[0] = 1;
			c[1].value[1] = floor(cos(theta*PI / 180));
			c[1].value[2] = floor(sin(theta*PI / 180));
			c[2].value[1] = floor(-sin(theta*PI / 180));
			c[2].value[2] = floor(cos(theta*PI / 180));

			if (zCoor == Divisable1) // 15
			{
				if (yCoor == Divisable1) // 15
				{
					b[3].value[0] = -xCoor;
					b[3].value[1] = -yCoor;
					b[3].value[2] = -zCoor - 1;
				}
				else if (yCoor == Divisable) // 16
				{
					b[3].value[0] = -xCoor;
					b[3].value[1] = -yCoor + 1;
					b[3].value[2] = -zCoor;
				}
			}
			else if (zCoor == Divisable) // 16
			{
				if (yCoor == Divisable1) // 15
				{
					b[3].value[0] = -xCoor;
					b[3].value[1] = -yCoor - 1;
					b[3].value[2] = -zCoor;
				}
				else if (yCoor == Divisable) // 16
				{
					b[3].value[0] = -xCoor;
					b[3].value[1] = -yCoor;
					b[3].value[2] = -zCoor + 1;
				}
			}
		}
		else if (axis == 'y')
		{
			c[1].value[1] = 1;
			c[0].value[0] = floor(cos(theta*PI / 180));
			c[0].value[2] = floor(-sin(theta*PI / 180));
			c[2].value[0] = floor(sin(theta*PI / 180));
			c[2].value[2] = floor(cos(theta*PI / 180));

			if (xCoor == Divisable1) // 15
			{
				if (zCoor == Divisable1) // 15
				{
					b[3].value[0] = -xCoor - 1;
					b[3].value[1] = -yCoor;
					b[3].value[2] = -zCoor;
				}
				else if (zCoor == Divisable) // 16
				{
					b[3].value[0] = -xCoor;
					b[3].value[1] = -yCoor;
					b[3].value[2] = -zCoor + 1;
				}
			}
			else if (xCoor == Divisable) // 16
			{
				if (zCoor == Divisable1) // 15
				{
					b[3].value[0] = -xCoor;
					b[3].value[1] = -yCoor;
					b[3].value[2] = -zCoor - 1;
				}
				else if (zCoor == Divisable) // 16
				{
					b[3].value[0] = -xCoor + 1;
					b[3].value[1] = -yCoor;
					b[3].value[2] = -zCoor;
				}
			}
		}
		else if (axis == 'z')
		{
			c[2].value[2] = 1;
			c[0].value[0] = floor(cos(theta*PI / 180));
			c[0].value[1] = floor(sin(theta*PI / 180));
			c[1].value[0] = floor(-sin(theta*PI / 180));
			c[1].value[1] = floor(cos(theta*PI / 180));

			if (xCoor == Divisable1) // 15
			{
				if (yCoor == Divisable1) // 15
				{
					b[3].value[0] = -xCoor;
					b[3].value[1] = -yCoor - 1;
					b[3].value[2] = -zCoor;
				}
				else if (yCoor == Divisable) // 16
				{
					b[3].value[0] = -xCoor - 1;
					b[3].value[1] = -yCoor;
					b[3].value[2] = -zCoor;
				}
			}
			else if (xCoor == Divisable) // 16
			{
				if (yCoor == Divisable1) // 15
				{
					b[3].value[0] = -xCoor + 1;
					b[3].value[1] = -yCoor;
					b[3].value[2] = -zCoor;
				}
				else if (yCoor == Divisable) // 16
				{
					b[3].value[0] = -xCoor;
					b[3].value[1] = -yCoor + 1;
					b[3].value[2] = -zCoor;
				}
			}
		}
	}

	cout << "Translation Matrix X Rotation Matrix: ";
	cout << endl;
	multiply4x4(a, c, result1);
	cout << "Rotation Matrix X Translation Matrix: ";
	cout << endl;
	multiply4x4(result1, b, result2);
	delete[](a);
	delete[](b);
	delete[](c);
	delete[](result1);
	return result2;
}

class Matrix
{
public:
	int numDivX = 32;
	int numDivY = 32;
	int numDivZ = 32;
	int voxelDataSize = numDivX * numDivY * numDivZ;
	float *voxelValue;
	float *TvoxelValue;
	Float4 *Coor;
	const char *fName = "toilet_0444.raw"; // Input .raw file
	void read();
	void save();
	void rotate(int xDist, int yDist, int zDist, char rotateAxis, int thetas);
};

// Reads .raw file & create corresponding coordinate matrix for voxels
void Matrix::read()
{
	size_t size = voxelDataSize*sizeof(float);

	FILE *fp = fopen(fName, "rb");

	if (!fp)
	{
		fprintf(stderr, "Error opening file '%s'\n", fName);
		abort();
	}

	unsigned char *tempdata = new unsigned char[voxelDataSize];
	size_t read = fread(tempdata, sizeof(unsigned char), voxelDataSize, fp);
	fclose(fp);
	printf("Read '%s', %d bytes\n", fName, read);
	this->voxelValue = new float[voxelDataSize];
	for (int k = 0; k < voxelDataSize; k++)
	{
		this->voxelValue[k] = float(ceil(tempdata[k] / 254));
		//cout << voxelValue[k] << " " << endl;
	}

	this->Coor = new Float4[voxelDataSize];
	for (int i = 0; i < numDivZ; i++)
	{
		for (int j = 0; j < numDivY; j++)
		{
			for (int k = 0; k < numDivX; k++)
			{
				int marker = (numDivZ*numDivZ*i) + (numDivY*j) + k;
				Coor[marker].value[0] = k;
				Coor[marker].value[1] = j;
				Coor[marker].value[2] = i;
				Coor[marker].value[3] = 1;
			}
		}
	}
	delete[] tempdata;

	//cout << "Original voxel values: " << endl;
	//for (int p = 0; p < voxelDataSize; p++)
	//{
	//	cout << voxelValue[p] << " ";
	//}
	//cout << endl;
}

// Perform complete voxel rotation in 3D space
void Matrix::rotate(int xDist, int yDist, int zDist, char rotateAxis, int thetas)
{
	Float4 *Transformed = new Float4[voxelDataSize]; // Final transformed matrix stored here
	TvoxelValue = new float[voxelDataSize]; // Final voxel value stored here
	Float4 *TransMat = gen4x4tm(xDist, yDist, zDist, rotateAxis, thetas); // Obtain transformation matrix
	multiplynxm(TransMat, Coor, Transformed, 4, voxelDataSize); // matrix multiply to get Transformed
	for (int fin = 0; fin < voxelDataSize; fin++) // rotate voxelsx
	{
		int yes = (Transformed[fin].value[2] * numDivZ * numDivZ) + (Transformed[fin].value[1] * numDivY) + Transformed[fin].value[0];
		this->TvoxelValue[yes] = voxelValue[fin];
	}
	delete[](Transformed);
	delete[](TransMat);

	//cout << "Transformed voxel values: " << endl;
	//for (int p = 0; p < voxelDataSize; p++)
	//{
	//	cout << TvoxelValue[p] << " ";
	//}
	//cout << endl;
}

// Saves rotated voxels back into raw file
void Matrix::save()
{
	ofstream rawFile;
	string fName = "phi_grid.raw";
	rawFile.open(fName, std::ofstream::binary);
	if (!rawFile.good())
	{
		cerr << "Unable to open output file for writing" << endl;
		abort();
	}
	char* phiOut = new char[voxelDataSize];
	for (int k = 0; k < voxelDataSize; k++)
	{
		//if (GPU)
		//	phiOut[k] = char(phiValGPU[k] * 255);
		//else
		phiOut[k] = char(TvoxelValue[k] * 255);
	}
	rawFile.write((char*)phiOut, voxelDataSize*sizeof(char));
	delete[] phiOut;
	rawFile.close();
	cout << "Voxel File Saved as : " << fName << endl << endl;
}

class Artificial
{
public:
	Float4 *place;
	int Xs = 3;
	int Ys = 3;
	int Zs = 3;
	int vSize = Xs * Ys * Zs;
	float *voxValue;
	float *TransformedVox;

	void address();
	void save();
	void rotate(int xDist, int yDist, int zDist, char rotateAxis, int thetas);
};


void Artificial::address()
{
	this->voxValue = new float[vSize]();
	// assign voxel value
	voxValue[0] = 1;
	voxValue[4] = 1;
	voxValue[5] = 1;
	voxValue[7] = 1;
	voxValue[13] = 1;
	voxValue[14] = 1;
	voxValue[16] = 1;
	voxValue[17] = 1;
	voxValue[22] = 1;


	cout << "Initialized voxel values: " << endl;
	for (int p = 0; p < vSize; p++)
	{
		cout << voxValue[p] << " ";
	}
	cout << endl;


	this->place = new Float4[vSize];
	for (int i = 0; i < Zs; i++)
	{
		for (int j = 0; j < Ys; j++)
		{
			for (int k = 0; k < Xs; k++)
			{
				int marker = (Zs*Zs*i) + (Ys*j) + k;
				place[marker].value[0] = k;
				place[marker].value[1] = j;
				place[marker].value[2] = i;
				place[marker].value[3] = 1;
			}
		}
	}
	cout << "Initial Location Matrix: " << endl;
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < vSize; j++) {
			cout << place[j].value[i] << " ";
		}
		cout << endl;
	}
	cout << endl;
}

void Artificial::rotate(int xDist, int yDist, int zDist, char rotateAxis, int thetas)
{
	Float4 *FinalT = new Float4[4]; // Final transformed matrix stored here
	TransformedVox = new float[vSize]; // Final voxel value stored here
	Float4 *TMat = gen4x4tm(xDist, yDist, zDist, rotateAxis, thetas); // Obtain transformation matrix
	multiplynxm(TMat, place, FinalT, 4, vSize); // matrix multiply to get Transformed
	for (int fin = 0; fin < vSize; fin++) // rotate voxels
	{
		int yes = (FinalT[fin].value[2] * Zs * Zs) + (FinalT[fin].value[1] * Ys) + FinalT[fin].value[0];
		this->TransformedVox[yes] = voxValue[fin];
	}
	delete[](FinalT);
	delete[](TMat);
	cout << "Transformed voxel values: " << endl;
	for (int p = 0; p < vSize; p++)
	{
		cout << TransformedVox[p] << " ";
	}
	cout << endl;
}

void Artificial::save()
{
	ofstream rawFile;
	string fName = "aiyo.raw";
	rawFile.open(fName, std::ofstream::binary);
	if (!rawFile.good())
	{
		cerr << "Unable to open output file for writing" << endl;
		abort();
	}
	char* phiOut = new char[vSize];
	for (int k = 0; k < vSize; k++)
	{
		//if (GPU)
		//	phiOut[k] = outputPhi(phiValGPU[k] * 255);
		//else
		//phiOut[k] = char(TransformedVox[k] * 255);
		phiOut[k] = char(voxValue[k] * 255);
	}
	rawFile.write((char*)phiOut, vSize*sizeof(char));
	delete[] phiOut;
	rawFile.close();
	cout << "Voxel File Saved as : " << fName << endl << endl;
}

int main()
{
	//Matrix wow;
	//wow.read();
	//wow.rotate(16, 16, 16, 'y', 90);
	//wow.save();

	Artificial Testz;
	Testz.address();
	Testz.rotate(1, 1, 1, 'y', 90);
	Testz.save();


	return 0;
}